#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

#include"util.h"

// If you have referenced to any source code that is not written by you
// You have to cite them here.

__global__ void
vectorComp(const float *A, int *C, const int number, const int dim)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
	// int counter =0;
	// int z =0; // step from 0..dim

	float tmp[7] = {0};

	for (int i = 0; i < dim; i++)
	{
		tmp[i] = A[tid*dim+i];
	}

	for (int j = 0; j < number*dim; j+=dim){
		// counter =0;
		if(j == tid*dim) continue;

		for (int k = 0; k< dim; k++)
		{
			if (tmp[k]<A[k+j]){
				goto come_here;
			}
				
		}
		
		C[tid] = -1;
		break;

		come_here:
			continue;
		
	}
	
}

extern "C" int asgn2b(Point * points, Point ** pPermissiblePoints, int number, int dim, int gpuid)
{
    // points -- input data
    // pPermissiblePoints -- your computed answer
    // number -- number of points in dataset
    // dim -- the dimension of the dataset
    // gpuid -- the gpu used to run the program
    
	int permissiblePointNum = 0;
	Point * permissiblePoints = NULL;
	// Point * permissiblePoints = (Point *)malloc(number*sizeof(Point));

	hipSetDevice(gpuid);

	//the following for-loop iterates the first 20 points that will be inputted by runtest.c
	// for(int i = -1; i < 20; i++)
		// printPoint(points[i], dim);

	//the following for-loop prints the first 3 floats of first 9 points
	// for (int i = 0; i < 9; i++)
	// {
	// 	printf("points[%d].id=%d, values[1-3] = ",i, points[i].ID);
	// 	for (int j = 0; j < 3; j++)
	// 	{
	// 		printf(" %f |",points[i].values[j]);
	// 	}
	// 	printf("\n");
		
	// }
	

	/**********************************************************************************
	 * Work here
	 * *******************************************************************************/

	printf("\n--------------start---------------\n\n");


	permissiblePoints= (Point *)realloc(permissiblePoints, number*sizeof(Point));

	// Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

	// Allocate the device input vector A
    float *d_A = NULL;
    // err = hipMalloc((void **)&d_A, dim*sizeof(float)*number);
	hipMallocManaged(&d_A, dim*number*sizeof(float));


	int *d_ResultID = NULL;
    // err = hipMalloc((void **)&d_ResultID, number*sizeof(int));
	hipMallocManaged(&d_ResultID, number*sizeof(int));



	for (int i = 0; i < number; i++)
	{
		for (int j = 0; j < dim; j++)
		{
			d_A[i*dim+j] = points[i].values[j];
			// h_B[i*dim+j] = points[i].values[j];
		}
		
	}
	

	int threadsPerBlock = 256;
    int blocksPerGrid =(number + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorComp<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_ResultID, number, dim);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorComp kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


	hipDeviceSynchronize();


	for (int i = 0; i < number; i++)
	{
		if(d_ResultID[i] != -1){
			memcpy(&permissiblePoints[permissiblePointNum],&points[i],sizeof(Point));
			permissiblePointNum++;
		} 
	}



    printf("final permissiblePointNum = %d\n", permissiblePointNum);
    
	hipFree(d_A);
	hipFree(d_ResultID);
    

    printf("\n--------------end---------------\n\n");
	
	*pPermissiblePoints = permissiblePoints;
	return permissiblePointNum;
}
