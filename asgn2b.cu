#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

#include"util.h"

// If you have referenced to any source code that is not written by you
// You have to cite them here.


__global__ void
vectorComp(const float *A, int *C, int number, int dim)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
	// if(tid>number) printf("tid = %d !!!!!!\n",tid);
	int counter =0;
	int z =0; // step from 0..dim
	int flag =0;
	
	for (int j = 0; j < number*dim; j++)
	{
		if (A[tid*dim+z] > A[j]){
			counter++;
			flag =1;
		}else if(A[tid*dim+z] == A[j]) 
			counter++;
		z++;
		if( z == dim ){

			if(flag && counter==dim){
				C[tid] = -1;
				break;
			} 
			counter=0;
			flag =0;
			z=0;
		}
		
	}
}

extern "C" int asgn2b(Point * points, Point ** pPermissiblePoints, int number, int dim, int gpuid)
{
    // points -- input data
    // pPermissiblePoints -- your computed answer
    // number -- number of points in dataset
    // dim -- the dimension of the dataset
    // gpuid -- the gpu used to run the program
    
	int permissiblePointNum = 0;
	Point * permissiblePoints = NULL;
	// Point * permissiblePoints = (Point *)malloc(number*sizeof(Point));

	hipSetDevice(gpuid);

	//the following for-loop iterates the first 20 points that will be inputted by runtest.c
	// for(int i = -1; i < 20; i++)
		// printPoint(points[i], dim);

	//the following for-loop prints the first 3 floats of first 9 points
	// for (int i = 0; i < 9; i++)
	// {
	// 	printf("points[%d].id=%d, values[1-3] = ",i, points[i].ID);
	// 	for (int j = 0; j < 3; j++)
	// 	{
	// 		printf(" %f |",points[i].values[j]);
	// 	}
	// 	printf("\n");
		
	// }
	

	/**********************************************************************************
	 * Work here
	 * *******************************************************************************/

	printf("\n--------------start---------------\n\n");


	permissiblePoints= (Point *)realloc(permissiblePoints, number*sizeof(Point));

	// Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

	// Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, dim*sizeof(float)*number);

	// Allocate the device input vector B
    // float *d_B = NULL;
    // err = hipMalloc((void **)&d_B, dim*sizeof(float)*number);


	int *d_ResultID = NULL;
    err = hipMalloc((void **)&d_ResultID, number*sizeof(int));


	float* h_A = (float*)malloc(sizeof(float)*dim*number);
	// float* h_B = (float*)malloc(sizeof(float)*dim*number);

	for (int i = 0; i < number; i++)
	{
		for (int j = 0; j < dim; j++)
		{
			h_A[i*dim+j] = points[i].values[j];
			// h_B[i*dim+j] = points[i].values[j];
		}
		
	}
	

	err = hipMemcpy(d_A, h_A, dim*sizeof(float)*number, hipMemcpyHostToDevice);


	// err = hipMemcpy(d_B, h_B, dim*sizeof(float)*number, hipMemcpyHostToDevice);


	int threadsPerBlock = 256;
    int blocksPerGrid =(number + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorComp<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_ResultID, number, dim);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorComp kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	int *h_ResultID = (int *)malloc(number*sizeof(int));
	err = hipMemcpy(h_ResultID, d_ResultID, number*sizeof(int), hipMemcpyDeviceToHost);

	
	//prints permissible points with ID<20 
	// for (int i = 0; i < 20; i++)
	// {
	// 	// printf("h_ResultID[%d]= %d | ",i,h_ResultID[i]);
	// 	if(h_ResultID[i] == 0)
	// 		printf("ID %d\n", i+1);
	// }
	// printf("\n");

	for (int i = 0; i < number; i++)
	{
		if(h_ResultID[i] != -1){
			memcpy(&permissiblePoints[permissiblePointNum],&points[i],sizeof(Point));
			permissiblePointNum++;
		} 
	}



    printf("final permissiblePointNum = %d\n", permissiblePointNum);
    
	hipFree(d_A);
	// hipFree(d_B);
	hipFree(d_ResultID);
	free(h_A);
	// free(h_B);
	free(h_ResultID);
    

    printf("\n--------------end---------------\n\n");
	
	*pPermissiblePoints = permissiblePoints;
	return permissiblePointNum;
}
