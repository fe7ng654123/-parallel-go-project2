#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

#include"util.h"

// If you have referenced to any source code that is not written by you
// You have to cite them here.




extern "C" int asgn2b(Point * points, Point ** pPermissiblePoints, int number, int dim, int gpuid)
{
    // points -- input data
    // pPermissiblePoints -- your computed answer
    // number -- number of points in dataset
    // dim -- the dimension of the dataset
    // gpuid -- the gpu used to run the program
    
	int permissiblePointNum = 0;
	Point * permissiblePoints = NULL;

	hipSetDevice(gpuid);

	//the following for-loop iterates the first 20 points that will be inputted by runtest.c
    //	for(int i = -1; i < 20; i++)
    //		printPoint(points[i], dim);

	/**********************************************************************************
	 * Work here
	 * *******************************************************************************/

	
	*pPermissiblePoints = permissiblePoints;
	return permissiblePointNum;
}
